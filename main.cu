#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstring>

#define FILE_PATH "primes.bin"
#define TEXT_FILE_PATH "primes_converted.txt"
#define THREADS_PER_BLOCK 256
#define SEGMENT_SIZE (1 << 22)

__device__ inline long long cuda_max(long long a, long long b) {
    return (a > b) ? a : b;
}

__global__ void sieve_kernel(char* d_is_prime, long long low, long long high, int* d_primes, int num_primes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < num_primes; i += blockDim.x * gridDim.x) {
        int p = d_primes[i];
        long long p_long = (long long)p;
        long long start = cuda_max(p_long * p_long, ((low + p_long - 1) / p_long) * p_long);
        if (start > high) continue;
        for (long long j = start; j <= high; j += p_long) {
            d_is_prime[j - low] = 0;
        }
    }
}

std::vector<int> simple_sieve(int limit) {
    int sqrt_limit = static_cast<int>(std::sqrt(limit)) + 1;
    std::vector<char> is_prime(sqrt_limit + 1, 1);
    is_prime[0] = is_prime[1] = 0;
    for (int p = 3; p * p <= sqrt_limit; p += 2) {
        if (is_prime[p]) {
            for (int i = p * p; i <= sqrt_limit; i += 2 * p) {
                is_prime[i] = 0;
            }
        }
    }
    std::vector<int> primes = {2};
    for (int p = 3; p <= sqrt_limit; p += 2) {
        if (is_prime[p]) {
            primes.push_back(p);
        }
    }
    return primes;
}

long long get_last_prime() {
    std::ifstream infile(FILE_PATH, std::ios::in | std::ios::binary | std::ios::ate);
    if (!infile.is_open()) {
        return 1;
    }
    std::streampos size = infile.tellg();
    if (size == 0) {
        infile.close();
        return 1;
    }
    infile.seekg(0, std::ios::beg);
    long long last_prime = 0;
    long long delta;
    while (infile.read(reinterpret_cast<char*>(&delta), sizeof(delta))) {
        last_prime += delta;
    }
    infile.close();
    return last_prime;
}

void bulk_save_primes(const std::vector<long long>& primes) {
    std::ofstream outfile(FILE_PATH, std::ios::app | std::ios::binary);
    if (!outfile.is_open()) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return;
    }
    static long long last_prime = get_last_prime();
    for (const auto& prime : primes) {
        long long delta = prime - last_prime;
        outfile.write(reinterpret_cast<const char*>(&delta), sizeof(delta));
        last_prime = prime;
    }
    outfile.close();
}

void read_and_convert() {
    std::ifstream infile(FILE_PATH, std::ios::in | std::ios::binary);
    if (!infile.is_open()) {
        std::cerr << "Error opening binary file for reading!" << std::endl;
        return;
    }
    std::ofstream outfile(TEXT_FILE_PATH, std::ios::out);
    if (!outfile.is_open()) {
        std::cerr << "Error opening text file for writing!" << std::endl;
        infile.close();
        return;
    }
    long long last_prime = 1;
    long long delta;
    while (infile.read(reinterpret_cast<char*>(&delta), sizeof(delta))) {
        last_prime += delta;
        outfile << last_prime << " ";
    }
    infile.close();
    outfile.close();
    std::cout << "Conversion complete. Primes written to " << TEXT_FILE_PATH << std::endl;
}

int main(int argc, char* argv[]) {
    bool read_mode = false;
    int batch_limit = -1;
    for (int i = 1; i < argc; ++i) {
        if (std::strcmp(argv[i], "--read") == 0) {
            read_mode = true;
        } else if (std::strcmp(argv[i], "--limit") == 0) {
            if (i + 1 < argc) {
                batch_limit = std::stoi(argv[i + 1]);
                ++i;
            } else {
                std::cerr << "Error: --limit requires a value" << std::endl;
                return 1;
            }
        }
    }
    if (read_mode) {
        read_and_convert();
        return 0;
    }
    long long current = get_last_prime() + 1;
    long long total_primes = 0;
    int max_digits = 0;
    int batch_counter = 0;
    bool running = true;
    auto start_time = std::chrono::steady_clock::now();
    while (running) {
        auto batch_start_time = std::chrono::steady_clock::now();
        long long high = current + SEGMENT_SIZE - 1;
        int sqrt_high = static_cast<int>(std::sqrt(high)) + 1;
        std::vector<int> primes = simple_sieve(sqrt_high);
        char* d_is_prime;
        int* d_primes;
        int num_primes = primes.size();
        hipMalloc((void**)&d_is_prime, SEGMENT_SIZE * sizeof(char));
        hipMalloc((void**)&d_primes, num_primes * sizeof(int));
        hipMemcpy(d_primes, primes.data(), num_primes * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_is_prime, 1, SEGMENT_SIZE * sizeof(char));
        int threadsPerBlock = THREADS_PER_BLOCK;
        int blocks = (num_primes + threadsPerBlock - 1) / threadsPerBlock;
        sieve_kernel<<<blocks, threadsPerBlock>>>(d_is_prime, current, high, d_primes, num_primes);
        hipDeviceSynchronize();
        std::vector<char> h_is_prime(SEGMENT_SIZE);
        hipMemcpy(h_is_prime.data(), d_is_prime, SEGMENT_SIZE * sizeof(char), hipMemcpyDeviceToHost);
        std::vector<long long> segment_primes;
        for (long long i = 0; i < SEGMENT_SIZE; ++i) {
            long long num = current + i;
            if (num < 2) continue;
            if (h_is_prime[i]) {
                segment_primes.push_back(num);
            }
        }
        long long primes_found_in_batch = segment_primes.size();
        total_primes += primes_found_in_batch;
        if (!segment_primes.empty()) {
            long long longest_prime = segment_primes.back();
            int digits = std::to_string(longest_prime).length();
            if (digits > max_digits) {
                max_digits = digits;
            }
            bulk_save_primes(segment_primes);
        }
        hipFree(d_is_prime);
        hipFree(d_primes);
        batch_counter++;
        auto batch_end_time = std::chrono::steady_clock::now();
        std::chrono::duration<double> batch_runtime = batch_end_time - batch_start_time;
        std::cout << "Batch " << batch_counter << ": Found " << primes_found_in_batch
                  << " primes. | Total Primes: " << total_primes
                  << " | Longest Prime Digits: " << max_digits
                  << " | Batch Runtime: " << batch_runtime.count() << " seconds" << std::endl;
        if (batch_limit > 0 && batch_counter >= batch_limit) {
            running = false;
        }
        current = high + 1;
    }
    auto total_runtime = std::chrono::steady_clock::now() - start_time;
    std::cout << "\nTotal Runtime: " << std::chrono::duration_cast<std::chrono::seconds>(total_runtime).count()
              << " seconds" << std::endl;
    std::cout << "Total Numbers Calculated: " << batch_counter * SEGMENT_SIZE << std::endl;
    std::cout << "Total Primes Found: " << total_primes << std::endl;
    return 0;
}