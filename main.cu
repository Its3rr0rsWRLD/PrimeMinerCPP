#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstring>
#include <stdint.h>

#define FILE_PATH "primes.bin"
#define TEXT_FILE_PATH "primes_converted.txt"
#define THREADS_PER_BLOCK 256
#define SEGMENT_SIZE (1 << 22)

__device__ inline long long cuda_max(long long a, long long b) {
    return (a > b) ? a : b;
}

__global__ void sieve_kernel(uint8_t* d_is_prime, long long current, long long high, int* d_primes, long long* d_offsets, int num_primes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int i = idx; i < num_primes; i += total_threads) {
        int p = d_primes[i];
        long long start = d_offsets[i];
        if (start == -1 || start > high) continue;
        long long p_long = (long long)p;
        for (long long j = start; j <= high; j += 2 * p_long) {
            long long index = (j - current) / 2;
            int byte_index = index / 8;
            int bit_index = index % 8;
            d_is_prime[byte_index] &= ~(1 << bit_index);
        }
    }
}

std::vector<int> simple_sieve(int limit) {
    int sqrt_limit = static_cast<int>(std::sqrt(limit)) + 1;
    std::vector<char> is_prime(sqrt_limit + 1, 1);
    is_prime[0] = is_prime[1] = 0;
    for (int p = 3; p * p <= sqrt_limit; p += 2) {
        if (is_prime[p]) {
            for (int i = p * p; i <= sqrt_limit; i += 2 * p) {
                is_prime[i] = 0;
            }
        }
    }
    std::vector<int> primes = {2};
    for (int p = 3; p <= sqrt_limit; p += 2) {
        if (is_prime[p]) {
            primes.push_back(p);
        }
    }
    return primes;
}

long long get_last_prime() {
    std::ifstream infile(FILE_PATH, std::ios::in | std::ios::binary | std::ios::ate);
    if (!infile.is_open()) {
        return 1;
    }
    std::streampos size = infile.tellg();
    if (size == 0) {
        infile.close();
        return 1;
    }
    infile.seekg(0, std::ios::beg);
    long long last_prime = 0;
    long long delta;
    while (infile.read(reinterpret_cast<char*>(&delta), sizeof(delta))) {
        last_prime += delta;
    }
    infile.close();
    return last_prime;
}

void bulk_save_primes(const std::vector<long long>& primes) {
    static long long last_prime = get_last_prime();
    std::ofstream outfile(FILE_PATH, std::ios::app | std::ios::binary);
    if (!outfile.is_open()) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return;
    }
    for (const auto& prime : primes) {
        long long delta = prime - last_prime;
        outfile.write(reinterpret_cast<const char*>(&delta), sizeof(delta));
        last_prime = prime;
    }
    outfile.close();
}

void read_and_convert() {
    std::ifstream infile(FILE_PATH, std::ios::in | std::ios::binary);
    if (!infile.is_open()) {
        std::cerr << "Error opening binary file for reading!" << std::endl;
        return;
    }
    std::ofstream outfile(TEXT_FILE_PATH, std::ios::out);
    if (!outfile.is_open()) {
        std::cerr << "Error opening text file for writing!" << std::endl;
        infile.close();
        return;
    }
    long long last_prime = 1;
    long long delta;
    while (infile.read(reinterpret_cast<char*>(&delta), sizeof(delta))) {
        last_prime += delta;
        outfile << last_prime << " ";
    }
    infile.close();
    outfile.close();
    std::cout << "Conversion complete. Primes written to " << TEXT_FILE_PATH << std::endl;
}

int main(int argc, char* argv[]) {
    bool read_mode = false;
    int batch_limit = -1;
    for (int i = 1; i < argc; ++i) {
        if (std::strcmp(argv[i], "--read") == 0) {
            read_mode = true;
        } else if (std::strcmp(argv[i], "--limit") == 0) {
            if (i + 1 < argc) {
                batch_limit = std::stoi(argv[i + 1]);
                ++i;
            } else {
                std::cerr << "Error: --limit requires a value" << std::endl;
                return 1;
            }
        }
    }
    if (read_mode) {
        read_and_convert();
        return 0;
    }
    long long current = get_last_prime() + 1;
    if (current % 2 == 0) current++;
    long long total_primes = 0;
    int max_digits = 0;
    int batch_counter = 0;
    bool running = true;
    auto start_time = std::chrono::steady_clock::now();

    uint8_t* d_is_prime;
    int* d_primes = nullptr;
    long long* d_offsets = nullptr;
    std::vector<uint8_t> h_is_prime((SEGMENT_SIZE + 7) / 8);

    while (running) {
        auto batch_start_time = std::chrono::steady_clock::now();
        long long high = current + 2 * (SEGMENT_SIZE - 1);
        int sqrt_high = static_cast<int>(std::sqrt(high)) + 1;
        std::vector<int> primes = simple_sieve(sqrt_high);
        int num_primes = primes.size();

        std::vector<long long> offsets(num_primes);
        for (int i = 0; i < num_primes; ++i) {
            int p = primes[i];
            if (p == 2) {
                offsets[i] = -1;
                continue;
            }
            long long p_long = (long long)p;
            long long start = (current + p_long - 1) / p_long * p_long;
            if (start % 2 == 0) start += p_long;
            if (start < p_long * p_long) start = p_long * p_long;
            if (start < current) start += 2 * p_long;
            offsets[i] = start;
        }

        static int max_num_primes = 0;
        if (num_primes > max_num_primes) {
            if (d_primes) hipFree(d_primes);
            if (d_offsets) hipFree(d_offsets);
            hipMalloc((void**)&d_primes, num_primes * sizeof(int));
            hipMalloc((void**)&d_offsets, num_primes * sizeof(long long));
            max_num_primes = num_primes;
        }

        hipMalloc((void**)&d_is_prime, h_is_prime.size() * sizeof(uint8_t));
        hipMemcpy(d_primes, primes.data(), num_primes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_offsets, offsets.data(), num_primes * sizeof(long long), hipMemcpyHostToDevice);
        hipMemset(d_is_prime, 0xFF, h_is_prime.size() * sizeof(uint8_t));

        int threadsPerBlock = THREADS_PER_BLOCK;
        int blocks = (num_primes + threadsPerBlock - 1) / threadsPerBlock;
        sieve_kernel<<<blocks, threadsPerBlock>>>(d_is_prime, current, high, d_primes, d_offsets, num_primes);
        hipDeviceSynchronize();

        hipMemcpy(h_is_prime.data(), d_is_prime, h_is_prime.size() * sizeof(uint8_t), hipMemcpyDeviceToHost);

        std::vector<long long> segment_primes;
        for (long long i = 0; i < SEGMENT_SIZE; ++i) {
            int byte_index = i / 8;
            int bit_index = i % 8;
            if (h_is_prime[byte_index] & (1 << bit_index)) {
                long long num = current + 2 * i;
                if (num >= 2) {
                    segment_primes.push_back(num);
                }
            }
        }

        long long primes_found_in_batch = segment_primes.size();
        total_primes += primes_found_in_batch;
        if (!segment_primes.empty()) {
            long long longest_prime = segment_primes.back();
            int digits = std::to_string(longest_prime).length();
            if (digits > max_digits) {
                max_digits = digits;
            }
            bulk_save_primes(segment_primes);
        }

        hipFree(d_is_prime);
        batch_counter++;
        auto batch_end_time = std::chrono::steady_clock::now();
        std::chrono::duration<double> batch_runtime = batch_end_time - batch_start_time;
        std::cout << "Batch " << batch_counter << ": Found " << primes_found_in_batch
                  << " primes. | Total Primes: " << total_primes
                  << " | Longest Prime Digits: " << max_digits
                  << " | Batch Runtime: " << batch_runtime.count() << " seconds" << std::endl;
        if (batch_limit > 0 && batch_counter >= batch_limit) {
            running = false;
        }
        current = high + 2;
    }

    hipFree(d_primes);
    hipFree(d_offsets);

    auto total_runtime = std::chrono::steady_clock::now() - start_time;
    std::cout << "\nTotal Runtime: " << std::chrono::duration_cast<std::chrono::seconds>(total_runtime).count()
              << " seconds" << std::endl;
    std::cout << "Total Numbers Calculated: " << batch_counter * SEGMENT_SIZE << std::endl;
    std::cout << "Total Primes Found: " << total_primes << std::endl;
    return 0;
}